#include "hip/hip_runtime.h"
/*
1. Create two VBO, one for position one for velocity of particles
2. Create two VBO, one for triangles drawing one for edge drawing
3. Register the VBOs with Cuda
4. Map the VBO for writing from Cuda
5. Run Cuda kernel to modify the vertex positions
6. Unmap the VBO
7. Render the results using OpenGL
*/

#include"Header.h"
// includes, cuda
#include <cuda_gl_interop.h>
#include <hiprand/hiprand_kernel.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop
#include <hip/hip_runtime.h>
#include <cub/hipcub/hipcub.hpp>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms


/* This is the name of the data file we will read. */
std::string modelfile = "56realport.ncdf";//"mod0.mod"
std::string  fieldfile = "56realport.mod";
const char* FILE_NAME = modelfile.c_str();
const char* FILE_NAME2 = fieldfile.c_str();

/* Handle errors by printing an error message and exiting with a
* non-zero status. */
#define ERRCODE 2
#define ERR(e) {printf("Error: %s\n", nc_strerror(e)); /*exit(ERRCODE);*/}


////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 1024;
const unsigned int window_height = 1024;
clock_t timer0;
int N_par = 1;
int N_cycles = 2;//number of RF cycles we are going to track the particles
double fre;
double Vc = 299792458.0;
double epsilon = 8.854187817e-12;
double mu = 1.2566370614e-6;
double me = 9.1e-31;
double qe = 1.60217657e-19;


//Variables
int isGPU = 1;
std::map<std::string, std::string> inputs;//inputs from input file
double3 Efield, Bfield;//field info on every nodes
double3 *Efd_temp, *Bfd_temp;
double3 *dEfd, *dBfd;
double3 *D_p_Efd, *D_p_Bfd;//store the field at the location of particle
double3* impactmomentum;
double3* dimpactmomentum;
double* impactenergy;
double* impactenergy_shuffle;
double* D_impactenergy;

hipError_t err;
double xrange[2] = { 0.117, 0.179 }, yrange[2] = { -0.19, -0.103 }, zrange[2] = { 1.574, 1.74 };

double fdnorm = 1e6;//normalize of field 
double fdnorm_max = 1e8;
double fdnorm_step = 1e6;
double initenergy = 2;//initial energy of seconday electrons, eV
double *Efd, *Bfd;//field info from file
double *Efd_img, *Bfd_img;

int* impact, *impact_shuffle;//number of impact
int* dimpact;


int* flag, *flag_shuffle;//indicate whether the particle is dead or live. 0 means live, 1 means just experinced an impact, -2 means dead, 
int* dflag;
double dt = 0.01;
int* initphase,*initphase_shuffle;
int* dinitphase;
int phase_step = 2;

double3 *Hposition, *Hvelocity;//position and momentum stored in host
double3 *Hposition_shuffle, *Hvelocity_shuffle;//position and momentum used for shuffle the dead particle out
double3 *d_position0, *d_momentumf;//store the intermediant position and momentum of particle during the tracking
double3 *H_position0, *H_momentumf, *H_position0_shuffle, *H_momentumf_shuffle;
double3 *D_momentumt;//store the intermediant momentum for Runge-Kutta steps
double4 *barycentric, *barycentric_shuffle;//the barycentric coordinates of the particle in each tet;four compoments are representing the coordinate coefficients corresponding to 4,1,2,3 vertex.
double4 *dbarycentric;
double *nodes;
double *dnodes;//nodes info for device use
double *mincor, *maxcor;
double *nodesdisp;//nodes for display
double *volume;//store the signed volume of the tets;
double *dvolume;

int *tetext;//node structure of exterior tets
int *tetint;//node structure of internal tets
int *tetall;//node structure of all tets, to be used in GPU
int *dtetall;
int4 *D_p_nodes;//store the temp info of the nodes of tet where the particle is located

int *meshindextet,*meshindextet_shuffle;// tet mesh, use for straight first then flush it with tet mesh info
int *meshindextet_temp;//use to store the active tetmesh temporarily, then we move the info to meshindextet
int *dmeshindextet;

int numactive;//the number of tet that is active(in the xyz range)


int *indexes;//the array stores the indexes of external tets
int *indexesedge;//the array stores the indexes of external surfaces of external tets.
unsigned int ntriangles;//number of triangles of exterior tetrahedrons
unsigned int nnodes, next,nint,nall;//number of total nodes, and number of exterior and interior tets


/*Generate the tet node structure, 14 entries:
first entry is the volume group id,
following four are node indexes,
next four are surface type(-1 means shared with neighbor, and other numbers represent the sideset id),
next one is the id where the tet is located in a easier to search straight mesh, the ID is calculated as sizeofx*sizeofy*Z+sizeofx*Y+X
next four are the neighbors cooresponding to the four sides. 
*/
int tetsize = 14;

double norm; //the normalize factor

struct diminfo
{
	char namedim[MAX_NC_NAME];//allocatet the array for dimension names
	size_t lengthp;//Length of the dimension
};

struct varinfo
{
	char namevar[MAX_NC_NAME];//name of the variable
	//#define	NC_NAT 	        0	/**< Not A Type */
	//#define	NC_BYTE         1	/**< signed 1 byte integer */
	//#define	NC_CHAR 	2	/**< ISO/ASCII character */
	//#define	NC_SHORT 	3	/**< signed 2 byte integer */
	//#define	NC_INT 	        4	/**< signed 4 byte integer */
	//#define NC_LONG         NC_INT  /**< deprecated, but required for backward compatibility. */
	//#define	NC_FLOAT 	5	/**< single precision floating point number */
	//#define	NC_DOUBLE 	6	/**< double precision floating point number */
	//#define	NC_UBYTE 	7	/**< unsigned 1 byte int */
	//#define	NC_USHORT 	8	/**< unsigned 2-byte int */
	//#define	NC_UINT 	9	/**< unsigned 4-byte int */
	//#define	NC_INT64 	10	/**< signed 8-byte int */
	//#define	NC_UINT64 	11	/**< unsigned 8-byte int */
	//#define	NC_STRING 	12	/**< string */
	nc_type typevar;//type of the variable
	int numdim;
	int *dimids = new int[NC_MAX_VAR_DIMS];//the id of dimensions that are used by this variable
	int numatt;//number of attributions
};

struct variablevalue
{
	int typevar;
	std::vector<int> value4;
	std::vector<double> value6;
};
int retval;//error code

// vbo variables
GLuint vbo, vbov;/*vbo is position, vbov is velocity*/
GLuint nvbo;//the vbo of nodes
GLuint vboindex;/*vboindex is the index of vertex*/
GLuint vboindexedg;/*vbo object for index of node to generate edges*/

struct hipGraphicsResource *cuda_vbo_resource, *cuda_vbov_resource;
void *d_vbo_buffer = NULL;
void *d_vbov_buffer = NULL;

double g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = -90.0;
float translate_z = -0.1;
float translate_x = 0;
float translate_y = 0.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;
const char *sSDKsample = "simpleGL (VBO)";

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
	unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource, struct hipGraphicsResource **vbov_resource, int func);
void runAutoTest(int devID, char **argv, char *ref_file);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);


/*Define functions*/
void tetvolumeHost(double3 point1, double3 point2, double3 point3, double3 point4, double *volume);
void ConnectTetHost_2(int* oldmesh, int tetsize, int nall, int nnodes);
void showcudaerror();
double3 p_functionHost(double3 position, double3 momentum, double3 Efield, double3 Bfield, double t, double fre, double phi0);
void intersectHost(double3 point1, double3 point2, double3 point3, double3 pointA, double3 pointB, double *pointintpara);
void alpha_betaHost(double3 point1, double3 point2, double3 point3, double3 point, double2 *ab);
void tetvolumeHost(double3 point1, double3 point2, double3 point3, double3 point4, double *volume);
void moveKernelHost(double4 *barycentric, double3 *position, double3 *momentum, double3* d_position0, double3* d_momentumf, double3* Efield, double3 *Bfield, double3* impactmomentum,
	double *nodes, double* volume,
	double norm, double t, double dt, double fre, double fdnorm,
	int* impact, int *meshindextet, int* tetmesh, int* flag, int* initphase,
	int tetsize, int N_par, int N_cycles, int phase_step);
void trackingHost(double4 *barycentric, double3 *positionold, double3 *positionnew, double3 *momentum, double3* d_momentumf, double3* impactmomentum,
	double norm, double *nodes, double *volume,
	int *meshindextet, int *oldmesh, int* flag, int* impact,
	int N_par, int N_cycles, int tetsize);

//Kernels
//calculate the signed volume of tet with four arbitrary vertex points. 
__device__ void tetvolume(double3 point1, double3 point2, double3 point3, double3 point4, double *volume)
{
	*volume = (point4.x - point1.x)*((point2.y - point1.y)*(point3.z - point1.z) - (point3.y - point1.y)*(point2.z - point1.z)) -
		(point4.y - point1.y)*((point2.x - point1.x)*(point3.z - point1.z) - (point3.x - point1.x)*(point2.z - point1.z)) +
		(point4.z - point1.z)*((point2.x - point1.x)*(point3.y - point1.y) - (point3.x - point1.x)*(point2.y - point1.y));
}
//calculate the intercept point of a plane(from three points) and a line(two points)
__device__ void intersect(double3 point1, double3 point2, double3 point3, double3 pointA, double3 pointB, double *pointintpara)
{
	double a, b, c, d;//parameter of plane

	double ndotu;
	/*for plane*/
	a = (point2.y - point1.y) * (point3.z - point1.z) - (point3.y - point1.y)*(point2.z - point1.z);
	b = -((point2.x - point1.x) * (point3.z - point1.z) - (point3.x - point1.x)*(point2.z - point1.z));
	c = (point2.x - point1.x) * (point3.y - point1.y) - (point3.x - point1.x)*(point2.y - point1.y);
	d = -a*point1.x - b*point1.y - c*point1.z;
	/*to find the t of the intersection point */
	//http://geomalgorithms.com/a05-_intersect-1.html
	ndotu = a*(pointB.x - pointA.x) + b*(pointB.y - pointA.y) + c*(pointB.z - pointA.z);
	if (ndotu != 0)
	{
		*pointintpara = -(a*pointA.x + b*pointA.y + c*pointA.z + d) / ndotu;
	}
	else
		*pointintpara = 1e6;
}
//find the alpha beta coordinate of intersection point on a triangle
__device__ void alpha_beta(double3 point1, double3 point2, double3 point3, double3 point, double2 *ab)
{
	/*Needed some trick to do the determint, assumed a fake third vector that gives {1,2,3} in the third row in parameter matrix, corresponding to a fake gama variable*/
	(*ab).x = ((point.x - point1.x)*((point3.y - point1.y) * 3 - (point3.z - point1.z) * 2) - (point3.x - point1.x)*((point.y - point1.y) * 3 - (point.z - point1.z) * 2) + 1 * ((point.y - point1.y)*(point3.z - point1.z) - (point.z - point1.z)*(point3.y - point1.y))) /
		((point2.x - point1.x)*((point3.y - point1.y) * 3 - (point3.z - point1.z) * 2) - (point3.x - point1.x)*((point2.y - point1.y) * 3 - (point2.z - point1.z) * 2) + 1 * ((point2.y - point1.y)*(point3.z - point1.z) - (point2.z - point1.z)*(point3.y - point1.y)));
	(*ab).y = ((point2.x - point1.x)*((point.y - point1.y) * 3 - (point.z - point1.z) * 2) - (point.x - point1.x)*((point2.y - point1.y) * 3 - (point2.z - point1.z) * 2) + 1 * ((point2.y - point1.y)*(point.z - point1.z) - (point2.z - point1.z)*(point.y - point1.y))) /
		((point2.x - point1.x)*((point3.y - point1.y) * 3 - (point3.z - point1.z) * 2) - (point3.x - point1.x)*((point2.y - point1.y) * 3 - (point2.z - point1.z) * 2) + 1 * ((point2.y - point1.y)*(point3.z - point1.z) - (point2.z - point1.z)*(point3.y - point1.y)));
}
//initailize the particles from the exterior surfaces
__global__ void initpar(double4 *barycentric, double3 *position, double3 *momentum, double3* d_position0, double3* d_momentumf, double3* D_momentumt,
	double* D_impactenergy,	double* nodes, double* volume,
	double norm,
	int* impact, int *meshindextet, int* flag, int* tetmesh, int*initphase,
	int N_par, int N_cycles, int tetsize, int phase_step)
{
	long int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j;//iteration index
	double3 vertexs[4];
	double a, b, c, d,tempvolume;
	int tempid;
	int tempmask[12] = { 0, 1, 2, 0, 2, 3, 0, 3, 2, 1, 2, 3 };

	/*Generate the particles on centers of the exterior triangles*/
	if (i<N_par)
	{
		tempid = meshindextet[i];
		for (j = 0; j < 4; j++)
		{
			vertexs[j].x = nodes[tetmesh[tempid*tetsize + 1 + j] * 3];
			vertexs[j].y = nodes[tetmesh[tempid*tetsize + 1 + j] * 3 + 1];
			vertexs[j].z = nodes[tetmesh[tempid*tetsize + 1 + j] * 3 + 2];
		}
		for (j = 0; j < 4; j++)
		{
			if (tetmesh[tempid*tetsize + 5 + j] != -1)
			{
				position[i].x = (vertexs[tempmask[j * 3]].x + vertexs[tempmask[j * 3 + 1]].x + vertexs[tempmask[j * 3 + 2]].x) / 3.0;
				position[i].y = (vertexs[tempmask[j * 3]].y + vertexs[tempmask[j * 3 + 1]].y + vertexs[tempmask[j * 3 + 2]].y) / 3.0;
				position[i].z = (vertexs[tempmask[j * 3]].z + vertexs[tempmask[j * 3 + 1]].z + vertexs[tempmask[j * 3 + 2]].z) / 3.0;

			}
		}
		d_position0[i].x = position[i].x;
		d_position0[i].y = position[i].y;
		d_position0[i].z = position[i].z;
		momentum[i].x = 0.0;
		momentum[i].y = 0.0;
		momentum[i].z = 0.0;
		d_momentumf[i].x = 0.0;
		d_momentumf[i].y = 0.0;
		d_momentumf[i].z = 0.0;
		D_momentumt[i].x = 0.0;
		D_momentumt[i].y = 0.0;
		D_momentumt[i].z = 0.0;
		impact[i] = 0;
		flag[i] = 0;
		for (j = 0; j < N_cycles*2; j++)
		{
			D_impactenergy[i*N_cycles*2 + j]= 0;
		}

		tempvolume = volume[tempid];
		tetvolume(vertexs[0], vertexs[1], vertexs[2], position[i], &a);
		barycentric[i].x = a / tempvolume;
		tetvolume(vertexs[2], vertexs[3], vertexs[0], position[i], &b);
		barycentric[i].y = b / tempvolume;
		tetvolume(vertexs[3], vertexs[1], vertexs[0], position[i], &c);
		barycentric[i].z = c / tempvolume;
		tetvolume(vertexs[1], vertexs[3], vertexs[2], position[i], &d);
		barycentric[i].w = d / tempvolume;

		position[i].x = position[i].x / norm;
		position[i].y = position[i].y / norm;
		position[i].z = position[i].z / norm;
		d_position0[i].x = d_position0[i].x;
		d_position0[i].y = d_position0[i].y;
		d_position0[i].z = d_position0[i].z;

		initphase[i] = i%phase_step;

		
	}
}
//locate the particle in tet mesh
__global__ void tracking(double4 *barycentric, double3 *positionold,  double3 *positionnew, double3 *momentum, double3* d_momentumf,
	double* D_impactenergy,double norm, double *nodes, double *volume,
	int *meshindextet, int *oldmesh, int* flag,  int* impact,
	int N_par, int N_cycles, int tetsize)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j;//iteration index.
	int tetindex;
	int count = 0;//if search for 10 tet and didn't find, the count as lost
	int tempflag;
	int found = 0;//flag indicating that we didn't find the tet that contain the particle yet.
	double tempvolume;
	double a, b, c, d;//the Barycentric coordinates of particle in tet;
	double interpara[4];//the parameters for intersections of trajectory with each face of tet
	double qe = 1.6e-19;
	double Vc = 299792458.0;
	double me = 9.10938291e-31;
	double2 ab[4];
	double3 nodePosition[4];//node position of one tet
	double3 intersec[4];//the intersection point on each plane of faces of a tet
	double3 pold, pnew;

	found = 0;
	if (i < N_par)
	{
		pold.x = positionold[i].x*norm;
		pold.y = positionold[i].y*norm;
		pold.z = positionold[i].z*norm;
		pnew.x = positionnew[i].x;
		pnew.y = positionnew[i].y;
		pnew.z = positionnew[i].z;
		tempflag = flag[i];
		tetindex = meshindextet[i];
		if (tempflag < -1)
		{
			return;
		}
		while (found == 0 && count <= 10)
		{
			count++;
			
			for (j = 0; j < 4; j++)
			{
				interpara[j] = 0;
				ab[j].x = 1e6;
				ab[j].y = 1e6;
			}
			//get the nodes of the tet that might contain the new position of particle, start from the old one
			for (j = 0; j < 4; j++)
			{
				nodePosition[j].x = nodes[oldmesh[tetindex*tetsize + 1 + j] * 3];
				nodePosition[j].y = nodes[oldmesh[tetindex*tetsize + 1 + j] * 3 + 1];
				nodePosition[j].z = nodes[oldmesh[tetindex*tetsize + 1 + j] * 3 + 2];
			}

			//get the signed volumes of tet, and calculate volume of 123P, 243P, 341P, 421P, in order to make the order right for surface recgonition we need to change the order 
			//012 cooresponding to face shared with first neighbor
			//123 cooresponding to face shared with fourth neighbor
			//230 cooresponding to face shared with second neighbor
			//310 cooresponding to face shared with third neighbor
			tempvolume = volume[tetindex];
			tetvolume(nodePosition[0], nodePosition[1], nodePosition[2], pnew, &a);
			a = a / tempvolume;
			tetvolume(nodePosition[2], nodePosition[3], nodePosition[0], pnew, &b);
			b = b / tempvolume;
			tetvolume(nodePosition[3], nodePosition[1], nodePosition[0], pnew, &c);
			c = c / tempvolume;
			tetvolume(nodePosition[1], nodePosition[3], nodePosition[2], pnew, &d);
			d = d / tempvolume;


			if (a >= -1e-15 && b >= -1e-15 && c >= -1e-15 && d >= -1e-15)
			{
				found = 1;
				barycentric[i].x = a;
				barycentric[i].y = b;
				barycentric[i].z = c;
				barycentric[i].w = d;
				tempflag ++;//record how long the particle has been tracked under the RF field, in unit of dt;
			}

			else
			{
				if (a < -1e-15)
				{
					intersect(nodePosition[0], nodePosition[1], nodePosition[2], pold, pnew, &interpara[0]);
					intersec[0].x = pold.x + interpara[0] * (pnew.x - pold.x);
					intersec[0].y = pold.y + interpara[0] * (pnew.y - pold.y);
					intersec[0].z = pold.z + interpara[0] * (pnew.z - pold.z);
					alpha_beta(nodePosition[0], nodePosition[1], nodePosition[2], intersec[0], &ab[0]);
				}
				if (b < -1e-15)
				{
					intersect(nodePosition[2], nodePosition[3], nodePosition[0], pold, pnew, &interpara[1]);
					intersec[1].x = pold.x + interpara[1] * (pnew.x - pold.x);
					intersec[1].y = pold.y + interpara[1] * (pnew.y - pold.y);
					intersec[1].z = pold.z + interpara[1] * (pnew.z - pold.z);
					alpha_beta(nodePosition[2], nodePosition[3], nodePosition[0], intersec[1], &ab[1]);
				}
				if (c < -1e-15)
				{
					intersect(nodePosition[3], nodePosition[1], nodePosition[0], pold, pnew, &interpara[2]);
					intersec[2].x = pold.x + interpara[2] * (pnew.x - pold.x);
					intersec[2].y = pold.y + interpara[2] * (pnew.y - pold.y);
					intersec[2].z = pold.z + interpara[2] * (pnew.z - pold.z);
					alpha_beta(nodePosition[3], nodePosition[1], nodePosition[0], intersec[2], &ab[2]);
				}
				if (d < -1e-15)
				{
					intersect(nodePosition[1], nodePosition[3], nodePosition[2], pold, pnew, &interpara[3]);
					intersec[3].x = pold.x + interpara[3] * (pnew.x - pold.x);
					intersec[3].y = pold.y + interpara[3] * (pnew.y - pold.y);
					intersec[3].z = pold.z + interpara[3] * (pnew.z - pold.z);
					alpha_beta(nodePosition[1], nodePosition[3], nodePosition[2], intersec[3], &ab[3]);
				}
				for (j = 0; j < 4; j++)
				{
					if (ab[j].x <= 1 && ab[j].x >= -1e-15 && ab[j].y <= 1 && ab[j].y > -1e-15 && ab[j].x + ab[j].y <= 1)
					{
						if (oldmesh[tetindex*tetsize + j + 5] != -1)//if the surface the particle hit is not a shared surface
						{
							if (tempflag > -1)//if the particle didn't hit a wall from previous time step
							{
								found = 1;
								barycentric[i].x = a > -1e-15 ? a : 0;
								barycentric[i].y = b > -1e-15 ? b : 0;
								barycentric[i].z = c > -1e-15 ? c : 0;
								barycentric[i].w = d > -1e-15 ? d : 0;
								tempflag = -1;//register it as "just hit a wall"

								D_impactenergy[i*N_cycles * 2 + impact[i]] = 
									Vc*(sqrt(d_momentumf[i].x*d_momentumf[i].x + d_momentumf[i].y*d_momentumf[i].y + d_momentumf[i].z*d_momentumf[i].z + me*me*Vc*Vc)-me*Vc)/qe;
								impact[i] ++;

								pnew.x = pold.x + interpara[j] * 0.5 * (pnew.x - pold.x);
								pnew.y = pold.y + interpara[j] * 0.5 * (pnew.y - pold.y);
								pnew.z = pold.z + interpara[j] * 0.5 * (pnew.z - pold.z);

								d_momentumf[i].x = 0;
								d_momentumf[i].y = 0;
								d_momentumf[i].z = 0;
								momentum[i].x = 0;
								momentum[i].y = 0;
								momentum[i].z = 0;
								j = 5;
							}
							else//if the particle hit a wall in last time step
							{
								found = 1;
								tempflag = -2;//means the particle is dead
								barycentric[i].x = 0;
								barycentric[i].y = 0;
								barycentric[i].z = 0;
								barycentric[i].w = 0;

								pnew.x = 0;
								pnew.y = 0;
								pnew.z = 0;
								d_momentumf[i].x = 0;
								d_momentumf[i].y = 0;
								d_momentumf[i].z = 0;
								momentum[i].x = 0;
								momentum[i].y = 0;
								momentum[i].z = 0;
								j = 5;
							}
						}
						else
						{
							tetindex = oldmesh[tetindex*tetsize + j + 10];//if particle hits a shared wall, then it moves to the neighbor tet
							j = 5;//jump out the for loop and start to search for tet again
						}
					}
				}
			}
			if (count == 10)
			{
				found = 1;
				tempflag = -3;//means the particle is lost
				barycentric[i].x = 0;
				barycentric[i].y = 0;
				barycentric[i].z = 0;
				barycentric[i].w = 0;

				pnew.x = 0;
				pnew.y = 0;
				pnew.z = 0;

				d_momentumf[i].x = 0;
				d_momentumf[i].y = 0;
				d_momentumf[i].z = 0;
				momentum[i].x = 0;
				momentum[i].y = 0;
				momentum[i].z = 0;
			}
		}
	}

	positionold[i].x = pnew.x / norm;
	positionold[i].y = pnew.y / norm;
	positionold[i].z = pnew.z / norm;
	positionnew[i].x = pnew.x ;
	positionnew[i].y = pnew.y ;
	positionnew[i].z = pnew.z ;
	momentum[i].x = d_momentumf[i].x;
	momentum[i].y = d_momentumf[i].y;
	momentum[i].z = d_momentumf[i].z;
	meshindextet[i] = tetindex;
	flag[i] = tempflag;
	
}

/*Update the momentum*/
__device__ double3 p_function(double3 momentum, double3 Efield, double3 Bfield, double t, double fre, double phi0)
{
	double3 dpodt;
	double qe = 1.6e-19;
	double Vc = 299792458.0;
	double me = 9.10938291e-31;
	double E = sqrt(momentum.x*momentum.x + momentum.y*momentum.y + momentum.z*momentum.z + me*me*Vc*Vc);
	double Phi = 2.0*M_PI*fre*t + phi0;
	double fsin, fcos;
	sincos(Phi, &fsin, &fcos);
	Efield.x = Efield.x*fsin;
	Efield.y = Efield.y*fsin;
	Efield.z = Efield.z*fsin;
	Bfield.x = -Bfield.x*fcos;
	Bfield.y = -Bfield.y*fcos;
	Bfield.z = -Bfield.z*fcos;

	dpodt.x = -qe*(Efield.x + Vc*(momentum.y*Bfield.z - momentum.z*Bfield.y)
		/ E);
	dpodt.y = -qe*(Efield.y + Vc*(-momentum.x*Bfield.z + momentum.z*Bfield.x)
		/ E);
	dpodt.z = -qe*(Efield.z + Vc*(momentum.x*Bfield.y - momentum.y*Bfield.x)
		/ E);
	return dpodt;
}
__global__ void getField(double4* D_barycentric, double3* D_p_Efd, double3* D_p_Bfd,  double3* Efield, double3* Bfield, double fdnorm,
	int* D_meshindextet, int* D_tetmesh, int N_par, int tetsize)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;

	double3 dfdtemp[4];
	double4 barycentrictemp;
	int j;
	int tetindex = D_meshindextet[i];
	if (i < N_par)
	{
		barycentrictemp.x = D_barycentric[i].x;
		barycentrictemp.y = D_barycentric[i].y;
		barycentrictemp.z = D_barycentric[i].z;
		barycentrictemp.w = D_barycentric[i].w;

		/*get the field for four vertex of the tet where the particle was found*/
		for (j = 0; j < 4; j++)
		{
			dfdtemp[j].x = Efield[D_tetmesh[tetindex*tetsize + j + 1]].x * fdnorm;
			dfdtemp[j].y = Efield[D_tetmesh[tetindex*tetsize + j + 1]].y * fdnorm;
			dfdtemp[j].z = Efield[D_tetmesh[tetindex*tetsize + j + 1]].z * fdnorm;
		}

		D_p_Efd[i].x = barycentrictemp.x *dfdtemp[3].x + barycentrictemp.y*dfdtemp[1].x + barycentrictemp.z*dfdtemp[2].x + barycentrictemp.w *dfdtemp[0].x;
		D_p_Efd[i].y = barycentrictemp.x *dfdtemp[3].y + barycentrictemp.y*dfdtemp[1].y + barycentrictemp.z*dfdtemp[2].y + barycentrictemp.w *dfdtemp[0].y;
		D_p_Efd[i].z = barycentrictemp.x *dfdtemp[3].z + barycentrictemp.y*dfdtemp[1].z + barycentrictemp.z*dfdtemp[2].z + barycentrictemp.w *dfdtemp[0].z;
		for (j = 0; j < 4; j++)
		{
			dfdtemp[j].x = Bfield[D_tetmesh[tetindex*tetsize + j + 1]].x * fdnorm;
			dfdtemp[j].y = Bfield[D_tetmesh[tetindex*tetsize + j + 1]].y * fdnorm;
			dfdtemp[j].z = Bfield[D_tetmesh[tetindex*tetsize + j + 1]].z * fdnorm;
		}
		D_p_Bfd[i].x = barycentrictemp.x *dfdtemp[3].x + barycentrictemp.y*dfdtemp[1].x + barycentrictemp.z*dfdtemp[2].x + barycentrictemp.w *dfdtemp[0].x;
		D_p_Bfd[i].y = barycentrictemp.x *dfdtemp[3].y + barycentrictemp.y*dfdtemp[1].y + barycentrictemp.z*dfdtemp[2].y + barycentrictemp.w *dfdtemp[0].y;
		D_p_Bfd[i].z = barycentrictemp.x *dfdtemp[3].z + barycentrictemp.y*dfdtemp[1].z + barycentrictemp.z*dfdtemp[2].z + barycentrictemp.w *dfdtemp[0].z;
	}
	
}
__global__ void rungeKutta(double3* D_positionold, double3* D_position0, double3* D_momentum0, double3* D_p_Efd, double3* D_p_Bfd,
	double t, double dt, double fre, double norm,
	int* D_initphase, int* flag,
	int N_par, int phase_step)
{
	t = t / fre;
	dt = dt / fre;
	double qe = 1.6e-19;
	double Vc = 299792458.0;
	double me = 9.10938291e-31;
	double E;
	double Phi;
	double fsin, fcos;
	int tempflag;
	double3 Efield, Bfield, Efield0, Bfield0;
	double3 dpodt;
	double3 momentum0, momentumt, momentumf;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N_par)
	{
		tempflag = flag[i];
		if (tempflag >= -1)
		{
			momentum0.x = D_momentum0[i].x;
			momentum0.y = D_momentum0[i].y;
			momentum0.z = D_momentum0[i].z;
			momentumf.x = momentum0.x;
			momentumf.y = momentum0.y;
			momentumf.z = momentum0.z;
			momentumt.x = momentum0.x;
			momentumt.y = momentum0.y;
			momentumt.z = momentum0.z;
			Efield0.x = D_p_Efd[i].x;
			Efield0.y = D_p_Efd[i].y;
			Efield0.z = D_p_Efd[i].z;
			Bfield0.x = -D_p_Bfd[i].x;
			Bfield0.y = -D_p_Bfd[i].y;
			Bfield0.z = -D_p_Bfd[i].z;
			/*First*/
			Phi = 2.0*M_PI*fre*t + (double)(D_initphase[i]) / (double)phase_step*M_PI;
			sincos(Phi, &fsin, &fcos);
			Efield.x = Efield0.x*fsin;
			Efield.y = Efield0.y*fsin;
			Efield.z = Efield0.z*fsin;
			Bfield.x = Bfield0.x*fcos;
			Bfield.y = Bfield0.y*fcos;
			Bfield.z = Bfield0.z*fcos;
			E = sqrt(momentumt.x*momentumt.x + momentumt.y*momentumt.y + momentumt.z*momentumt.z + me*me*Vc*Vc);
			dpodt.x = -qe*(Efield.x + Vc*(momentumt.y*Bfield.z - momentumt.z*Bfield.y)
				/ E);
			dpodt.y = -qe*(Efield.y + Vc*(-momentumt.x*Bfield.z + momentumt.z*Bfield.x)
				/ E);
			dpodt.z = -qe*(Efield.z + Vc*(momentumt.x*Bfield.y - momentumt.y*Bfield.x)
				/ E);
			momentumt.x = momentum0.x + 0.5 * dpodt.x*dt;
			momentumt.y = momentum0.y + 0.5 * dpodt.y*dt;
			momentumt.z = momentum0.z + 0.5 * dpodt.z*dt;
			momentumf.x += dpodt.x*dt / 6.0;
			momentumf.y += dpodt.y*dt / 6.0;
			momentumf.z += dpodt.z*dt / 6.0;
			///*Second*/
			//Phi = 2.0*M_PI*fre*(t + dt / 2.0) + (double)(D_initphase[i]) / (double)phase_step*M_PI;
			//sincos(Phi, &fsin, &fcos);
			//Efield.x = Efield0.x*fsin;
			//Efield.y = Efield0.y*fsin;
			//Efield.z = Efield0.z*fsin;
			//Bfield.x = Bfield0.x*fcos;
			//Bfield.y = Bfield0.y*fcos;
			//Bfield.z = Bfield0.z*fcos;
			//E = sqrt(momentumt.x*momentumt.x + momentumt.y*momentumt.y + momentumt.z*momentumt.z + me*me*Vc*Vc);
			//dpodt.x = -qe*(Efield.x + Vc*(momentumt.y*Bfield.z - momentumt.z*Bfield.y)
			//	/ E);
			//dpodt.y = -qe*(Efield.y + Vc*(-momentumt.x*Bfield.z + momentumt.z*Bfield.x)
			//	/ E);
			//dpodt.z = -qe*(Efield.z + Vc*(momentumt.x*Bfield.y - momentumt.y*Bfield.x)
			//	/ E);
			//momentumt.x = momentum0.x + 0.5 * dpodt.x*dt;
			//momentumt.y = momentum0.y + 0.5 * dpodt.y*dt;
			//momentumt.z = momentum0.z + 0.5 * dpodt.z*dt;
			//momentumf.x += dpodt.x*dt / 3.0;
			//momentumf.y += dpodt.y*dt / 3.0;
			//momentumf.z += dpodt.z*dt / 3.0;
			///*Third*/
			//Phi = 2.0*M_PI*fre*(t + dt / 2.0) + (double)(D_initphase[i]) / (double)phase_step*M_PI;
			//sincos(Phi, &fsin, &fcos);

			//Efield.x = Efield0.x*fsin;
			//Efield.y = Efield0.y*fsin;
			//Efield.z = Efield0.z*fsin;
			//Bfield.x = Bfield0.x*fcos;
			//Bfield.y = Bfield0.y*fcos;
			//Bfield.z = Bfield0.z*fcos;
			//E = sqrt(momentumt.x*momentumt.x + momentumt.y*momentumt.y + momentumt.z*momentumt.z + me*me*Vc*Vc);
			//dpodt.x = -qe*(Efield.x + Vc*(momentumt.y*Bfield.z - momentumt.z*Bfield.y)
			//	/ E);
			//dpodt.y = -qe*(Efield.y + Vc*(-momentumt.x*Bfield.z + momentumt.z*Bfield.x)
			//	/ E);
			//dpodt.z = -qe*(Efield.z + Vc*(momentumt.x*Bfield.y - momentumt.y*Bfield.x)
			//	/ E);
			//momentumt.x = momentum0.x + dpodt.x*dt;
			//momentumt.y = momentum0.y + dpodt.y*dt;
			//momentumt.z = momentum0.z + dpodt.z*dt;
			//momentumf.x += dpodt.x*dt / 3.0;
			//momentumf.y += dpodt.y*dt / 3.0;
			//momentumf.z += dpodt.z*dt / 3.0;
			///*Fourth*/
			//Phi = 2.0*M_PI*fre*(t + dt) + (double)(D_initphase[i]) / (double)phase_step*M_PI;
			//sincos(Phi, &fsin, &fcos);

			//Efield.x = Efield0.x*fsin;
			//Efield.y = Efield0.y*fsin;
			//Efield.z = Efield0.z*fsin;
			//Bfield.x = Bfield0.x*fcos;
			//Bfield.y = Bfield0.y*fcos;
			//Bfield.z = Bfield0.z*fcos;
			//E = sqrt(momentumt.x*momentumt.x + momentumt.y*momentumt.y + momentumt.z*momentumt.z + me*me*Vc*Vc);
			//dpodt.x = -qe*(Efield.x + Vc*(momentumt.y*Bfield.z - momentumt.z*Bfield.y)
			//	/ E);
			//dpodt.y = -qe*(Efield.y + Vc*(-momentumt.x*Bfield.z + momentumt.z*Bfield.x)
			//	/ E);
			//dpodt.z = -qe*(Efield.z + Vc*(momentumt.x*Bfield.y - momentumt.y*Bfield.x)
			//	/ E);
			//momentumf.x += dpodt.x*dt / 6.0;
			//momentumf.y += dpodt.y*dt / 6.0;
			//momentumf.z += dpodt.z*dt / 6.0;

			//E = sqrt(momentumf.x*momentumf.x + momentumf.y*momentumf.y + momentumf.z*momentumf.z + me*me*Vc*Vc);

			D_momentum0[i].x = momentumf.x;
			D_momentum0[i].y = momentumf.y;
			D_momentum0[i].z = momentumf.z;

			/*D_position0[i].x = D_positionold[i].x*norm + momentumf.x / E*Vc*dt;
			D_position0[i].y = D_positionold[i].y*norm + momentumf.y / E*Vc*dt;
			D_position0[i].z = D_positionold[i].z*norm + momentumf.z / E*Vc*dt;*/
		}
	}
}
/*update the position of partiles*/
__global__ void moveKernel(double3 *position, double3 *momentum, double3* d_position0, double3* d_momentumf, double3* D_p_Efd, double3* D_p_Bfd, 
	double norm, double t, double dt, double fre, 
	int* initphase, int* flag, 
	int N_par, int phase_step)
{
	
	double Vc = 299792458.0;
	double me = 9.10938291e-31;
	double phi0;

	double3 pEfd, pBfd;//fields at particle location
	double3 position0;
	double3 momentumt, momentum0, momentum1, momentum2, momentum3, momentum4, momentumf;
	double3 v;


	t = t / fre;
	dt = dt / fre;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j;
	int tempflag;
	if (i < N_par )
	{
		tempflag = flag[i];
		if (tempflag >= -1)
		{
			position0.x = position[i].x*norm;
			position0.y = position[i].y*norm;
			position0.z = position[i].z*norm;
			momentum0.x = momentum[i].x;
			momentum0.y = momentum[i].y;
			momentum0.z = momentum[i].z;
			
			pEfd.x = D_p_Efd[i].x;
			pEfd.y = D_p_Efd[i].y;
			pEfd.z = D_p_Efd[i].z;
			pBfd.x = D_p_Bfd[i].x;
			pBfd.y = D_p_Bfd[i].y;
			pBfd.z = D_p_Bfd[i].z;

			phi0 = (double)(initphase[i]) / phase_step*M_PI;

			//Runge-Kutta method to find the next momentum.
			momentumf = p_function(momentum0, pEfd, pBfd, t, fre, phi0);
			momentum1.x = momentumf.x*dt;
			momentum1.y = momentumf.y*dt;
			momentum1.z = momentumf.z*dt;

			momentumt.x = momentum0.x + momentum1.x / 2.0;
			momentumt.y = momentum0.y + momentum1.y / 2.0;
			momentumt.z = momentum0.z + momentum1.z / 2.0;
			momentumf = p_function(momentumt, pEfd, pBfd, t + dt / 2.0, fre, phi0);
			momentum2.x = momentumf.x*dt;
			momentum2.y = momentumf.y*dt;
			momentum2.z = momentumf.z*dt;

			momentumt.x = momentum0.x + momentum2.x / 2.0;
			momentumt.y = momentum0.y + momentum2.y / 2.0;
			momentumt.z = momentum0.z + momentum2.z / 2.0;
			momentumf = p_function(momentumt, pEfd, pBfd, t + dt / 2.0, fre, phi0);
			momentum3.x = momentumf.x*dt;
			momentum3.y = momentumf.y*dt;
			momentum3.z = momentumf.z*dt;

			momentumt.x = momentum0.x + momentum3.x;
			momentumt.y = momentum0.y + momentum3.y;
			momentumt.z = momentum0.z + momentum3.z;
			momentumf = p_function(momentumt, pEfd, pBfd, t + dt, fre, phi0);
			momentum4.x = momentumf.x*dt;
			momentum4.y = momentumf.y*dt;
			momentum4.z = momentumf.z*dt;

			momentumf.x = momentum0.x + momentum1.x / 6.0 + momentum2.x / 3.0 + momentum3.x / 3.0 + momentum4.x / 6.0;
			momentumf.y = momentum0.y + momentum1.y / 6.0 + momentum2.y / 3.0 + momentum3.y / 3.0 + momentum4.y / 6.0;
			momentumf.z = momentum0.z + momentum1.z / 6.0 + momentum2.z / 3.0 + momentum3.z / 3.0 + momentum4.z / 6.0;

			//Compute new velocity:
			v.x = Vc*momentumf.x / sqrt(me*me*Vc*Vc + momentumf.x*momentumf.x + momentumf.y*momentumf.y + momentumf.z*momentumf.z);
			v.y = Vc*momentumf.y / sqrt(me*me*Vc*Vc + momentumf.x*momentumf.x + momentumf.y*momentumf.y + momentumf.z*momentumf.z);
			v.z = Vc*momentumf.z / sqrt(me*me*Vc*Vc + momentumf.x*momentumf.x + momentumf.y*momentumf.y + momentumf.z*momentumf.z);
			//Compute new position;
			position0.x += v.x*dt;
			position0.y += v.y*dt;
			position0.z += v.z*dt;
			//put the temporary new position and momentum into global mem for later use. 
			d_position0[i].x = position0.x;
			d_position0[i].y = position0.y;
			d_position0[i].z = position0.z;

			d_momentumf[i].x = momentumf.x;
			d_momentumf[i].y = momentumf.y;
			d_momentumf[i].z = momentumf.z;

		}

	}
}
__global__ void dumpimpat(double3* impactengergy, int N_par);


void activetet(double* nodes, double* xrange, double* yrange, double* zrange, double norm, int* tetall, int* meshindextet_temp, int* N_par, int nall,int tetsize)
{
	/*Find the tet that is in the given box*/
	int i, j, k;
	double3 *centers;
	std::vector<int> tempactive;
	centers = new double3[nall];
	for (i = 0; i < nall; i++)
	{
		centers[i].x = (nodes[tetall[i*tetsize + 1] * 3] + nodes[tetall[i*tetsize + 2] * 3] + nodes[tetall[i*tetsize + 3] * 3] + nodes[tetall[i*tetsize + 4] * 3])/4;
		centers[i].y = (nodes[tetall[i*tetsize + 1] * 3 + 1] + nodes[tetall[i*tetsize + 2] * 3 + 1] + nodes[tetall[i*tetsize + 3] * 3 + 1] + nodes[tetall[i*tetsize + 4] * 3 + 1]) / 4 ;
		centers[i].z = (nodes[tetall[i*tetsize + 1] * 3 + 2] + nodes[tetall[i*tetsize + 2] * 3 + 2] + nodes[tetall[i*tetsize + 3] * 3 + 2] + nodes[tetall[i*tetsize + 4] * 3 + 2]) / 4 ;
	}
	for (i = 0; i < nall; i++)
	{
		if (centers[i].x > xrange[0] && centers[i].x<xrange[1] && centers[i].y>yrange[0] && centers[i].y<yrange[1] && centers[i].z>zrange[0] && centers[i].z < zrange[1]
			&& (tetall[i*tetsize + 5] != -1 || tetall[i*tetsize + 6] != -1 || tetall[i*tetsize + 7] != -1 || tetall[i*tetsize + 8] != -1))
		{
			tempactive.push_back(i);
		}
	}
	j = tempactive.size();
	for (i = 0; i < j; i++)
	{
		meshindextet_temp[i] = tempactive[i];
	}
	*N_par = j*phase_step;
	/*std::ofstream activefile("active.txt");
	for (i = 0; i < j; i++)
	{
		activefile << meshindextet_temp[i] << std::endl;
	}*/
	std::cout << "Activate tet finished, N_par = " <<*N_par<< std::endl;
}
void showcudaerror()
{
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cout << "CUDA Error:"<<hipGetErrorString(err)<<std::endl;
	}
}
void getinput()
{
	std::string line;
	std::string item;
	std::ifstream myfile;
	std::vector<std::string> elems;
	
	char delim;
	char waitchar;
	delim = ':';

	myfile.open("input.txt");
	if (myfile.is_open())
	{
		while (std::getline(myfile, line))
		{
			std::stringstream ss(line);
			while (std::getline(ss, item, delim))
			{
				elems.push_back(item);
			}
			if (elems.size() > 0)
			{
				elems[0].erase(std::remove_if(elems[0].begin(), elems[0].end(), isspace), elems[0].end());
				elems[1].erase(std::remove_if(elems[1].begin(), elems[1].end(), isspace), elems[1].end());
				elems[1].erase(std::remove(elems[1].begin(), elems[1].end(), ';'), elems[1].end());
				inputs[elems[0]] = elems[1];
				elems.clear();
			}
		}
		myfile.close();
		FILE_NAME = inputs["ModelFile"].c_str();
		FILE_NAME2 = inputs["FieldFile"].c_str();
		isGPU = stoi(inputs["isGPU"]);
		fdnorm = stod(inputs["FieldNorm_min"]);
		fdnorm_max = stod(inputs["FieldNorm_max"]);
		fdnorm_step = stod(inputs["FieldNorm_step"]);
		xrange[0] = stod(inputs["X_min"]);
		xrange[1] = stod(inputs["X_max"]);
		yrange[0] = stod(inputs["Y_min"]);
		yrange[1] = stod(inputs["Y_max"]);
		zrange[0] = stod(inputs["Z_min"]);
		zrange[1] = stod(inputs["Z_max"]);
		dt = stod(inputs["dt"]);
		phase_step = stod(inputs["Phase_sample_steps"]);
		N_cycles = stod(inputs["N_cyclse"]);
		initenergy = stod(inputs["Initial_Energy"]);
	}
	else
	{
		std::cout << "Couldn't find input.txt" << std::endl;
		exit(4);
	}

	
}
void shuffle(double4* barycentric, double3* position, double3* momentum, double* impactenergy, int* flag, int* meshindextet, int* impact,int* intiphase,
	double4* barycentric_shuffle, double3* position_shuffle, double3* momentum_shuffle, double* impactenergy_shuffle, int* flag_shuffle, int* meshindextet_shuffle, int* impact_shuffle, int* initphase_shuffle,int* N_par)
{
	int i,j;
	int countLive = 0;
	for (i = 0; i < *N_par; i++)
	{
		if ((flag[i]>-2)&&flag[i]<10/dt)//not dead due to lost or hitting the wall, and not slowly flying in the field for too long (>10 T)
		{
			barycentric_shuffle[countLive] = barycentric[i];
			position_shuffle[countLive] = position[i];
			momentum_shuffle[countLive] = momentum[i];
			flag_shuffle[countLive] = flag[i];
			meshindextet_shuffle[countLive] = meshindextet[i];
			impact_shuffle[countLive] = impact[i];
			initphase_shuffle[countLive] = initphase[i];
			memcpy((void*)(impactenergy_shuffle+countLive*N_cycles * 2), (void*)(impactenergy+i*N_cycles * 2), N_cycles * 2*sizeof(double));
			countLive++;
		}
	}
	*N_par = countLive;
}
void launch_kernel(double3 *pos, double3 *vel, double frametime, int func)
{
	int i,j;
	// execute the kernel
	dim3 block(256, 1, 1);
	dim3 grid(N_par/block.x+1,1,1);
	hipError_t err;
	if (func == 0)
	{
		int* dindex;//indexes for exterior triangles for device
		hipMalloc(&dindex, ntriangles*sizeof(int)*3);
		hipMemcpy(dindex, indexes, ntriangles * sizeof(int)*3, hipMemcpyHostToDevice);

		hipMalloc(&dnodes, nnodes*sizeof(double)*3);
		hipMemcpy(dnodes, nodes, nnodes * sizeof(double)*3, hipMemcpyHostToDevice);
		
		//initialize the barycentric coordinates of the particles to zeros;
		
		for (i = 0; i < N_par; i++)
		{
			barycentric[i].w = 0;
			barycentric[i].x = 0;
			barycentric[i].y = 0;
			barycentric[i].z = 0;
		}
		
		hipMemcpy(dbarycentric, barycentric, N_par*sizeof(double4), hipMemcpyHostToDevice);
		//activetet(nodes, xrange, yrange, zrange, norm, tetall, meshindextet, &N_par, nall, tetsize);
		for (i = 0; i < N_par / phase_step; i++)//change the tet index every phase_step numbers in meshindextet 
		{
			for (j = 0; j < phase_step; j++)
			{
				meshindextet[i*phase_step + j] = meshindextet_temp[i];
			}
		}
		hipMemcpy(dmeshindextet, meshindextet, N_par*sizeof(int), hipMemcpyHostToDevice);
		/*initialize the particle positions and momentum*/
		initpar << <grid, block >> >(dbarycentric, pos, vel, d_position0, d_momentumf, D_momentumt, D_impactenergy, dnodes, dvolume, norm, dimpact, dmeshindextet, dflag, dtetall, dinitphase, N_par, N_cycles, tetsize, phase_step);
		
		if (isGPU == 0)
		{
			hipMemcpy(barycentric, dbarycentric, N_par * sizeof(double4), hipMemcpyDeviceToHost);
			hipMemcpy(Efd_temp, dEfd, nnodes * sizeof(double3), hipMemcpyDeviceToHost);
			hipMemcpy(Bfd_temp, dBfd, nnodes * sizeof(double3), hipMemcpyDeviceToHost);
			hipMemcpy(meshindextet, dmeshindextet, sizeof(int)*N_par, hipMemcpyDeviceToHost);
			hipMemcpy(impact, dimpact, sizeof(int)*N_par, hipMemcpyDeviceToHost);
			hipMemcpy(impactenergy, D_impactenergy, sizeof(double)*N_par*N_cycles * 2, hipMemcpyDeviceToHost);
			hipMemcpy(flag, dflag, sizeof(int)*N_par, hipMemcpyDeviceToHost);
			hipMemcpy(Hposition, pos, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
			hipMemcpy(Hvelocity, vel, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
			hipMemcpy(H_position0, d_position0, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
			hipMemcpy(H_momentumf, d_momentumf, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
			hipMemcpy(initphase, dinitphase, sizeof(int)*N_par, hipMemcpyDeviceToHost);
		}
		hipMemcpy(barycentric, dbarycentric, N_par * sizeof(double4), hipMemcpyDeviceToHost);
		hipMemcpy(Efd_temp, dEfd, nnodes * sizeof(double3), hipMemcpyDeviceToHost);
		hipMemcpy(Bfd_temp, dBfd, nnodes * sizeof(double3), hipMemcpyDeviceToHost);
		hipMemcpy(meshindextet, dmeshindextet, sizeof(int)*N_par, hipMemcpyDeviceToHost);
		hipMemcpy(impact, dimpact, sizeof(int)*N_par, hipMemcpyDeviceToHost);
		hipMemcpy(impactenergy, D_impactenergy, sizeof(double)*N_par*N_cycles * 2, hipMemcpyDeviceToHost);
		hipMemcpy(flag, dflag, sizeof(int)*N_par, hipMemcpyDeviceToHost);
		hipMemcpy(Hposition, pos, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
		hipMemcpy(Hvelocity, vel, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
		hipMemcpy(H_position0, d_position0, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
		hipMemcpy(H_momentumf, d_momentumf, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
		hipMemcpy(initphase, dinitphase, sizeof(int)*N_par, hipMemcpyDeviceToHost);
		std::cout << "Trying to initialize particles" << std::endl;
		printf("Field Normalizer = %2.3e  \n", fdnorm);
		printf("Initial Phase = %2.2f \n", g_fAnim);
		printf("dt = %2.3f*T \n", dt);

	}
	else
	{
		if (isGPU==1)
		{
			if (frametime>2&&(int)(frametime/dt)%(int)(2/dt) == 0)
			{
				hipMemcpy(Hposition, pos, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
				hipMemcpy(Hvelocity, vel, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
				hipMemcpy(flag, dflag, sizeof(int)*N_par, hipMemcpyDeviceToHost);
				hipMemcpy(impact, dimpact, sizeof(int)*N_par, hipMemcpyDeviceToHost);
				hipMemcpy(meshindextet, dmeshindextet, sizeof(int)*N_par, hipMemcpyDeviceToHost);
				hipMemcpy(barycentric, dbarycentric, sizeof(double4)*N_par, hipMemcpyDeviceToHost);
				hipMemcpy(impactenergy, D_impactenergy, sizeof(double)*N_par*N_cycles*2, hipMemcpyDeviceToHost);
				int countLive;//count the number of live particles
				int i;
				for (i = 0; i < N_par; i++)
				{
					countLive += (flag[i]>-2&&flag[i]<5/dt) ? 1 : 0;//if the flag of the particle is larger than -2, we count it as live.
				}
				/*get rid of the dead particles*/
				shuffle(barycentric, Hposition, Hvelocity, impactenergy, flag, meshindextet, impact,initphase,
					barycentric_shuffle, Hposition_shuffle, Hvelocity_shuffle, impactenergy_shuffle, flag_shuffle, meshindextet_shuffle, impact_shuffle, initphase_shuffle,&N_par);

				memcpy(barycentric, barycentric_shuffle, sizeof(double4)*N_par);
				memcpy(Hposition, Hposition_shuffle, sizeof(double3)*N_par);
				memcpy(Hvelocity, Hvelocity_shuffle, sizeof(double3)*N_par);
				memcpy(flag, flag_shuffle, sizeof(int)*N_par);
				memcpy(meshindextet, meshindextet_shuffle, sizeof(int)*N_par);
				memcpy(impact, impact_shuffle, sizeof(int)*N_par);
				memcpy(initphase, initphase_shuffle, sizeof(int)*N_par);
				memcpy(impactenergy, impactenergy_shuffle, sizeof(double)*N_par*N_cycles*2);
/*
				barycentric = barycentric_shuffle;
				Hposition = Hposition_shuffle;
				Hvelocity = Hvelocity_shuffle;
				flag = flag_shuffle;
				meshindextet = meshindextet_shuffle;
				impact = impact_shuffle;
				impactenergy = impactenergy_shuffle;*/

				hipMemcpy(pos, Hposition, sizeof(double3)*N_par, hipMemcpyHostToDevice);
				hipMemcpy(vel, Hvelocity, sizeof(double3)*N_par, hipMemcpyHostToDevice);
				hipMemcpy(dflag, flag, sizeof(int)*N_par, hipMemcpyHostToDevice);
				hipMemcpy(dimpact, impact, sizeof(int)*N_par, hipMemcpyHostToDevice);
				hipMemcpy(dinitphase, initphase, sizeof(int)*N_par, hipMemcpyHostToDevice);
				hipMemcpy(dmeshindextet, meshindextet, sizeof(int)*N_par, hipMemcpyHostToDevice);
				hipMemcpy(dbarycentric, barycentric, sizeof(double4)*N_par, hipMemcpyHostToDevice);
				hipMemcpy(D_impactenergy, impactenergy, sizeof(double)*N_par*N_cycles * 2, hipMemcpyHostToDevice);

			}

			/*Move the particle by one time step*/
			block.x = 128;
			grid.x = N_par / block.x + 1;
			getField <<< grid, block >>> (dbarycentric, D_p_Efd, D_p_Bfd, dEfd, dBfd, fdnorm,
				dmeshindextet, dtetall, N_par,tetsize);
			block.x = 64;
			grid.x = N_par / block.x + 1;
			//rungeKutta << < grid, block >> > (pos, d_position0, d_momentumf, D_p_Efd, D_p_Bfd, frametime, dt, fre, norm, dinitphase, dflag,N_par, phase_step);
			moveKernel << < grid, block >> >(pos, vel, d_position0, d_momentumf, D_p_Efd, D_p_Bfd, 
				norm, frametime, dt, fre, dinitphase, dflag, N_par, phase_step);
			/*Locate the particle*/
			block.x = 128;
			grid.x = N_par / block.x + 1;
			tracking << <grid, block >> >(dbarycentric, pos, d_position0, vel, d_momentumf, D_impactenergy, norm, dnodes, dvolume, dmeshindextet, dtetall, dflag, dimpact, N_par, N_cycles, tetsize);
			hipMemcpy(Hposition, pos, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
			hipMemcpy(flag, dflag, sizeof(int)*N_par, hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
		}
		
		else
		{
			timer0=clock();
			moveKernelHost(barycentric, Hposition, Hvelocity, H_position0, H_momentumf,Efd_temp, Bfd_temp, impactmomentum, 
				nodes, volume, 
				norm, frametime,dt, fre, fdnorm, 
				impact, meshindextet, tetall, flag, initphase,
				tetsize,N_par, N_cycles, phase_step);
			std::cout << "Time to run MoveKernel on CPU is " << ((double)clock() - (double)timer0) / CLOCKS_PER_SEC*1000.0 << " mS\n";

			timer0 = clock();
			trackingHost(barycentric, Hposition, H_position0, Hvelocity, H_momentumf, impactmomentum, norm, nodes, volume, meshindextet, tetall, flag, impact, N_par,
				N_cycles, tetsize);
			std::cout << "Time to run tracking on CPU is " << ((double)clock() - (double)timer0) / CLOCKS_PER_SEC*1000.0 << " mS\n";
			hipMemcpy(pos, Hposition, N_par * sizeof(double3), hipMemcpyHostToDevice);
			hipMemcpy(vel, Hvelocity, N_par * sizeof(double3), hipMemcpyHostToDevice);
		}
		

	}
}

bool checkHW(char *name, const char *gpuType, int dev)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	strcpy(name, deviceProp.name);

	if (!STRNCASECMP(deviceProp.name, gpuType, strlen(gpuType)))
	{
		return true;
	}
	else
	{
		return false;
	}
}

int findGraphicsGPU(char *name)
{
	int nGraphicsGPU = 0;
	int deviceCount = 0;
	bool bFoundGraphics = false;
	char firstGraphicsName[256], temp[256];

	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("> FAILED %s sample finished, exiting...\n", sSDKsample);
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("> There are no device(s) supporting CUDA\n");
		return false;
	}
	else
	{
		printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
	}

	for (int dev = 0; dev < deviceCount; ++dev)
	{
		bool bGraphics = !checkHW(temp, (const char *)"Tesla", dev);
		printf("> %s\t\tGPU %d: %s\n", (bGraphics ? "Graphics" : "Compute"), dev, temp);

		if (bGraphics)
		{
			if (!bFoundGraphics)
			{
				strcpy(firstGraphicsName, temp);
			}

			nGraphicsGPU++;
		}
	}

	if (nGraphicsGPU)
	{
		strcpy(name, firstGraphicsName);
	}
	else
	{
		strcpy(name, "this hardware");
	}

	return nGraphicsGPU;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

	// Get the number of processors in this system
	int iCPU = omp_get_num_procs();
	std::cout << iCPU << std::endl;
	// Now set the number of threads
	omp_set_num_threads(iCPU);
	char *ref_file = NULL;
	getinput();
	std::string exitchar;
	/* Loop indexes, and error handling. */
	int i = 0, j = 0, k =0, l = 0;
	/* Open the file. NC_NOWRITE tells netCDF we want read-only access
	* to the file.*/
	int ncid;
	if ((retval = nc_open(FILE_NAME, NC_NOWRITE, &ncid)))
	{
		ERR(retval);
		std::cout << std::endl;
		std::cout << "Press Enter to Exit";
		std::getline(std::cin, exitchar);
		exit(ERRCODE);
	}
	printf("The File ID is: %d\n", ncid);
	int tempid1, tempid2;//temporay variables to store id info of interested dimensions
	size_t templ1, templ2;//temporary variables to store dimension length info.

	/*read the nodes info*/
	std::cout << "Start to read the nodes info from file." << std::endl;
	if ((retval = nc_inq_dimid(ncid, "ncoords", &tempid1)))//get the id of the dimension which name is ncoords.(which represent the number of nodes)
		ERR(retval);
	if ((retval = nc_inq_dimlen(ncid, tempid1, &templ1)))//get the length of the dimension that represent the number of nodes.
		ERR(retval);
	nnodes = templ1;//total length of the coordinates 
	nodes = new double[nnodes * 3];//construct the array for the nodes.
	nodesdisp = new double[nnodes * 3];
	if ((retval = nc_inq_varid(ncid, "coords", &tempid1)))//get the id of the variable that represent the nodes.
		ERR(retval);
	if ((retval = nc_get_var_double(ncid, tempid1, nodes)))//read in the value of node coordinates
		ERR(retval);

	mincor = std::min_element(nodes, nodes + templ1 - 1);
	maxcor = std::max_element(nodes, nodes + templ1 - 1);
	norm = abs(*mincor) >= abs(*maxcor) ? abs(*mincor) : abs(*maxcor);

	for (i = 0; i < nnodes * 3; i++)
	{
		nodesdisp[i] = nodes[i] / norm;
	}

	std::cout << "Nodes info readin complete." << std::endl;
	std::cout << "Writing the nodes info to a file" << std::endl;
	/*write the nodes to a file*/
	std::ofstream nodefile("nodes.txt");
	for (i = 0; i < nnodes; i++)
	{
		nodefile << "ID: "<<i<<"  "<<nodes[i * 3] << " , " << nodes[i * 3 + 1] << " , " << nodes[i * 3 + 2] << std::endl;
	}
	nodefile.close();
	std::cout << "Nodes info writing complete!" << std::endl;

	/*read the extiror tet info*/
	std::cout << "Start to read the exterior tet info from file." << std::endl;
	if ((retval = nc_inq_dimid(ncid, "tetexterior", &tempid1)))//get the id of the dimension which name is tetinterior.(which represent the number of interior tetrahedron)
		ERR(retval);
	if ((retval = nc_inq_dimlen(ncid, tempid1, &templ1)))//get the length of the dimension that represent the interior tet.
		ERR(retval);
	next = templ1;
	tetext = new int[next * 9];
	if ((retval = nc_inq_varid(ncid, "tetrahedron_exterior", &tempid1)))//get the id of the variable that represent the exterior tetrahedron.
		ERR(retval);
	if ((retval = nc_get_var_int(ncid, tempid1, tetext)))
		ERR(retval);

	for (i = 0; i < next; i++)
	{
		for (j = 0; j < 4; j++)
		{
			if (tetext[i * 9 + j + 5] != -1)//-1 means a shared surface with other tet
			{
				ntriangles++;
			}
		}
	}
	std::cout << "Exterior Tet info readin complete." << std::endl;

	/*Make the index array for the drawing*/
	std::cout << "Constructing the index array for model drawing" << std::endl;
	indexes = new int[ntriangles * 3];
	indexesedge = new int[ntriangles * 6];
	int tempi = 0;//temporary index;
	int tempiedge = 0;
	for (i = 0; i < next; i++)
	{
		for (j = 0; j < 4; j++)
		{
			if (tetext[i * 9 + j + 5] != -1)//-1 means a shared surface with other tet
			{
				switch (j)
				{
				case 0:
					indexes[tempi] = tetext[i * 9 + 1];
					indexesedge[tempiedge] = tetext[i * 9 + 1];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 2];
					tempiedge++;
					tempi++;
					indexes[tempi] = tetext[i * 9 + 2];
					indexesedge[tempiedge] = tetext[i * 9 + 2];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 3];
					tempiedge++;
					tempi++;
					indexes[tempi] = tetext[i * 9 + 3];
					indexesedge[tempiedge] = tetext[i * 9 + 3];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 1];
					tempiedge++;
					tempi++;
					break;

				case 1:
					indexes[tempi] = tetext[i * 9 + 1];
					indexesedge[tempiedge] = tetext[i * 9 + 1];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 3];
					tempiedge++;
					tempi++;
					indexes[tempi] = tetext[i * 9 + 3];
					indexesedge[tempiedge] = tetext[i * 9 + 3];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 4];
					tempiedge++;
					tempi++;
					indexes[tempi] = tetext[i * 9 + 4];
					indexesedge[tempiedge] = tetext[i * 9 + 4];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 1];
					tempiedge++;
					tempi++;
					break;

				case 2:
					indexes[tempi] = tetext[i * 9 + 1];
					indexesedge[tempiedge] = tetext[i * 9 + 1];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 4];
					tempiedge++;
					tempi++;
					indexes[tempi] = tetext[i * 9 + 4];
					indexesedge[tempiedge] = tetext[i * 9 + 4];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 2];
					tempiedge++;
					tempi++;
					indexes[tempi] = tetext[i * 9 + 2];
					indexesedge[tempiedge] = tetext[i * 9 + 2];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 1];
					tempiedge++;
					tempi++;
					break;

				case 3:
					indexes[tempi] = tetext[i * 9 + 2];
					indexesedge[tempiedge] = tetext[i * 9 + 2];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 3];
					tempiedge++;
					tempi++;
					indexes[tempi] = tetext[i * 9 + 3];
					indexesedge[tempiedge] = tetext[i * 9 + 3];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 4];
					tempiedge++;
					tempi++;
					indexes[tempi] = tetext[i * 9 + 4];
					indexesedge[tempiedge] = tetext[i * 9 + 4];
					tempiedge++;
					indexesedge[tempiedge] = tetext[i * 9 + 2];
					tempiedge++;
					tempi++;
					break;

				}
			}
		}
	}
	i = 0; j = 0; k = 0;
	/*Write the index of exterior triangles info to a file*/
	std::ofstream trianglesfile("ext_Triangles.txt");
	for (i = 0; i < ntriangles; i++)
	{
		trianglesfile << "ID: " << i << "   " << indexes[i * 3] << " , " << indexes[i * 3 + 1] << " , " << indexes[i * 3 + 2] << std::endl;
	}
	trianglesfile.close();
	std::cout << "Index array for model drawing construction complete" << std::endl;

	/*read the interior tet info*/
	std::cout << "Reading interior tet info from file" << std::endl;
	if (!nc_inq_dimid(ncid, "tetinterior", &tempid2))//get the id of the dimension which name is tetinterior.(which represent the number of interior tetrahedron)
	{
		if ((retval = nc_inq_dimlen(ncid, tempid2, &templ2)))//get the length of the dimension that represent the interior tet.
			ERR(retval);
		nint = templ2;
		tetint = new int[nint * 5];//no need to specify the surface info here, so only 5 entries is needed, first one for material, following four for node id. 
		if ((retval = nc_inq_varid(ncid, "tetrahedron_interior", &tempid2)))//get the id of the variable that represent the exterior tetrahedron.
			ERR(retval);
		if ((retval = nc_get_var_int(ncid, tempid2, tetint)))
			ERR(retval);
		std::cout << "Interior tet info readin complete;" << std::endl;
	}
	else
	{
		std::cout << "None interior tet found" << std::endl;
	}

	/*Generate the tet node structure, 14 entries:
	first entry is the volume group id,
	following four are node indexes, 
	next four are surface type(-1 means shared with neighbor, and other numbers represent the sideset id),
	next one is the id where the tet is located in a easier to search straight mesh, the ID is calculated as sizeofx*sizeofy*Z+sizeofx*Y+X
	next four are neighbors
	*/
	nall = next + nint;
	//N_par = nall/4;
	tetall = new int[nall * tetsize];
	for (i = 0; i < nall*tetsize; i++)
	{
		tetall[i] = -1;
	}
	double3 tempnodes[4]; //the temp variable to store the nodes of the tet
	double *tempvolume;
	volume = (double*)malloc(sizeof(double)*nall);

	/*tempvolume = (double*)malloc(sizeof(double));
	*tempvolume = 0.0;
	tetvolumeHost({ -5.0, 5.0, -5.0}, { 5.0, 5.0, -5.0}, { 5.0, 5.0, 5.0 }, { 0.05, 0.3474, -0.226}, tempvolume);
	std::cout << *tempvolume << std::endl;*/
	
	for(i =0;i<next;i++)//move the data from tetext to tetall, since tetext has a period of 9 and tetall has a period of 13, we have to do some trick to put everything in line;
	{
		for (j = 0; j < 9; j++)
		{
			tetall[i*tetsize + j] = tetext[i * 9 + j];
			if (j>0 && j < 5)
			{
				tempnodes[j - 1].x = nodes[tetall[i*tetsize + j]*3];
				tempnodes[j - 1].y = nodes[tetall[i*tetsize + j]*3 + 1];
				tempnodes[j - 1].z = nodes[tetall[i*tetsize + j]*3 + 2];
			}
		}
		tetvolumeHost(tempnodes[0], tempnodes[1], tempnodes[2], tempnodes[3], (volume+i));
    }
	
	for (i = 0; i<nint; i++)//similarly for tetint
	{
		for (j = 0; j < 5; j++)
		{
			tetall[(i + next) * tetsize + j] = tetint[i * 5 + j];
			if (j>0 && j < 5)
			{
				tempnodes[j - 1].x = nodes[tetall[(i + next)*tetsize + j]*3];
				tempnodes[j - 1].y = nodes[tetall[(i + next)*tetsize + j]*3 + 1];
				tempnodes[j - 1].z = nodes[tetall[(i + next)*tetsize + j]*3 + 2];
			}
		}
		tetvolumeHost(tempnodes[0], tempnodes[1], tempnodes[2], tempnodes[3], volume+i+next);
	}
	

	/*Link the tet mesh together*/
	//ConnectTetHost(tetall, tetsize, nall);
	ConnectTetHost_2(tetall, tetsize, nall,nnodes);
	/*std::cout << "Tet mesh Linked!" << std::endl;
	std::ofstream meshfile("mesh.txt");
	for (i = 0; i < nall; i++)
	{
		meshfile << i << ":  ";
		for (j = 0; j < tetsize; j++)
			meshfile << tetall[i*tetsize + j] << ";";
		meshfile << std::endl;
	}*/
	/*send the volume info to device*/
	hipMalloc(&dvolume, sizeof(double)*nall);
	showcudaerror();
	hipMemcpy(dvolume, volume, sizeof(double)*nall, hipMemcpyHostToDevice);
	showcudaerror();
	/*send the tet mesh info to device*/
	hipMalloc(&dtetall, sizeof(int)*nall*tetsize);
	showcudaerror();
	hipMemcpy(dtetall, tetall, sizeof(int)*nall*tetsize, hipMemcpyHostToDevice);
	showcudaerror();
		
	/*Allocate the memory to store the info of where the partiles are located in tet mesh*/
	meshindextet_temp = (int *)malloc(sizeof(int)*nall);
	for (i = 0; i < nall; i++)
	{
		meshindextet_temp[i] = -1;
	}
	activetet(nodes, xrange, yrange, zrange, norm, tetall, meshindextet_temp, &N_par, nall, tetsize);

	meshindextet = (int *)malloc(sizeof(int)*N_par);
	meshindextet_shuffle = (int *)malloc(sizeof(int)*N_par);
	for (i = 0; i < N_par; i++)
	{
		meshindextet[i] = -1;
	}
	for (i = 0; i < N_par / phase_step; i++)//change the tet index every phase_step numbers in meshindextet 
	{
		for (j = 0; j < phase_step; j++)
		{
			meshindextet[i*phase_step + j] = meshindextet_temp[i];
		}
	}

	hipMalloc(&dmeshindextet, sizeof(int)*N_par);
	showcudaerror();
	hipMemcpy(dmeshindextet, meshindextet, sizeof(int)*N_par, hipMemcpyHostToDevice);
	showcudaerror();


	/*Read the field info from file*/
	std::cout << "Reading the field info from file2;" << std::endl;
	if ((retval = nc_open(FILE_NAME2, NC_NOWRITE, &ncid)))
	{
		ERR(retval);
		std::cout << std::endl;
		std::cout << "Press Enter to Exit";
		std::getline(std::cin, exitchar);
		exit(ERRCODE);
	}
	printf("The File ID is: %d\n", ncid);
	
	/*read the eField info*/
	Efd = new double[nnodes * 3];//construct the array for the Efield.
	Efd_img = new double[nnodes * 3];//construct the array for the imaginary part of Efield.
	for (i = 0; i < nnodes * 3; i++)
	{
		Efd[i] = 0;
		Efd_img[i] = 0;
	}
	if ((retval = nc_inq_varid(ncid, "efield", &tempid1)))//get the id of the variable that represent the efield.
	{
		ERR(retval);
	}
	else if ((retval = nc_get_var_double(ncid, tempid1, Efd)))//read in the value of efield on each node
	{
		ERR(retval);
	}
	if ((retval = nc_inq_varid(ncid, "efield_imag", &tempid1)))//get the id of the variable that represent the imaginary part of the efield.
	{
		ERR(retval);
	}
	else if ((retval = nc_get_var_double(ncid, tempid1, Efd_img)))//read in the value of efield on each node
	{
		ERR(retval);
	}

	/*read the bField info*/
	Bfd = new double[nnodes * 3];//construct the array for the Efield.
	Bfd_img = new double[nnodes * 3];//construct the array for the imaginary part of Efield.
	for (i = 0; i < nnodes * 3; i++)
	{
		Bfd[i] = 0;
		Bfd_img[i] = 0;
	}
	if ((retval = nc_inq_varid(ncid, "bfield", &tempid1)))//get the id of the variable that represent the bfield.
	{
		ERR(retval);
	}
	else if ((retval = nc_get_var_double(ncid, tempid1, Bfd)))//read in the value of bfield on each node
	{
		ERR(retval);
	}
	if ((retval = nc_inq_varid(ncid, "bfield_imag", &tempid1)))//get the id of the variable that represent the imaginary part of bfield.
	{
		ERR(retval);
	}
	else if ((retval = nc_get_var_double(ncid, tempid1, Bfd_img)))//read in the value of imag bfield
	{
		ERR(retval);
	}
	if ((retval = nc_inq_varid(ncid, "frequency", &tempid1)))//get the id of the variable that represent the frequency.
	{
		if ((retval = nc_inq_varid(ncid, "frequencyreal", &tempid1)))
		{
			ERR(retval);
		}
		else if ((retval = nc_get_var_double(ncid, tempid1, &fre)))//read in the value of imag bfield
		{
			ERR(retval);
		}
	}
	else if ((retval = nc_get_var_double(ncid, tempid1, &fre)))//read in the value of imag bfield
	{
		ERR(retval);
	}
	
	///*print field*/
	/*std::ofstream efieldfile("Efield.txt");
	for (i = 0; i < nnodes * 3; i++)
	{
		efieldfile<< Efd[i] << "	;	" << Efd_img[i] << std::endl;
	}
	efieldfile.close();
	std::ofstream bfieldfile("Bfield.txt");

	for (i = 0; i < nnodes * 3; i++)
	{
		bfieldfile << Bfd[i] << "	;	" << Bfd_img[i] << std::endl;
	}
	bfieldfile.close();*/
	
	/*put the field together so we don't need to copy four arraies of field to GPU, most of the case only one part is non_zero so we can do the following instead of finding the average*/
	Efd_temp = (double3 *)malloc(sizeof(double3)*nnodes);
	Bfd_temp = (double3 *)malloc(sizeof(double3)*nnodes);
	if (Bfd_img[0] == 0)
	{
		for (i = 0; i < nnodes; i++)
		{
			/*std::cout << Bfd[i * 3] <<" "<<Efd[i*3]<< std::endl;
			std::cout << Bfd_img[i * 3] << " " << Efd_img[i * 3] << std::endl;
			std::cout << Bfd[i * 3 + 1] << " " << Efd[i * 3+1] << std::endl;
			std::cout << Bfd_img[i * 3 + 1] << " " << Efd_img[i * 3 + 1] << std::endl;
			std::cout << Bfd[i * 3 + 2] << " " << Efd[i * 3 + 2] << std::endl;
			std::cout << Bfd_img[i * 3 + 2] << " " << Efd_img[i * 3 + 2] << std::endl;
			std::cout << std::endl;
			Efd_temp[i].x = sqrt(Efd[i * 3] * Efd[i * 3] + Efd_img[i * 3] * Efd_img[i * 3])*((Efd[i*3]>0)-(Efd[i*3]<0));
			Efd_temp[i].y = sqrt(Efd[i * 3 + 1] * Efd[i * 3 + 1] + Efd_img[i * 3 + 1] * Efd_img[i * 3 + 1]) * ((Efd[i * 3 + 1]>0) - (Efd[i * 3 + 1]<0));
			Efd_temp[i].z = sqrt(Efd[i * 3 + 2] * Efd[i * 3 + 2] + Efd_img[i * 3 + 2] * Efd_img[i * 3 + 2]) * ((Efd[i * 3 + 2]>0) - (Efd[i * 3 + 2]<0));
			Bfd_temp[i].x = sqrt(Bfd[i * 3] * Bfd[i * 3] + Bfd_img[i * 3] * Bfd_img[i * 3])*((Bfd[i * 3]>0) - (Bfd[i * 3]<0));
			Bfd_temp[i].y = sqrt(Bfd[i * 3 + 1] * Bfd[i * 3 + 1] + Bfd_img[i * 3 + 1] * Bfd_img[i * 3 + 1])*((Bfd[i * 3+1]>0) - (Bfd[i * 3+1]<0));
			Bfd_temp[i].z = sqrt(Bfd[i * 3 + 2] * Bfd[i * 3 + 2] + Bfd_img[i * 3 + 2] * Bfd_img[i * 3 + 2])*((Bfd[i * 3+2]>0) - (Bfd[i * 3+2]<0));
			*/
			Efd_temp[i].x = Efd[i * 3];
			Efd_temp[i].y = Efd[i * 3 + 1];
			Efd_temp[i].z = Efd[i * 3 + 2];
			Bfd_temp[i].x = Bfd[i * 3];
			Bfd_temp[i].y = Bfd[i * 3 + 1];
			Bfd_temp[i].z = Bfd[i * 3 + 2];
			//std::cout << Bfd_temp[i].x << std::endl;
		}
	}
	else
	{
		for (i = 0; i < nnodes; i++)
		{
			/*std::cout << Bfd[i * 3] <<" "<<Efd[i*3]<< std::endl;
			std::cout << Bfd_img[i * 3] << " " << Efd_img[i * 3] << std::endl;
			std::cout << Bfd[i * 3 + 1] << " " << Efd[i * 3+1] << std::endl;
			std::cout << Bfd_img[i * 3 + 1] << " " << Efd_img[i * 3 + 1] << std::endl;
			std::cout << Bfd[i * 3 + 2] << " " << Efd[i * 3 + 2] << std::endl;
			std::cout << Bfd_img[i * 3 + 2] << " " << Efd_img[i * 3 + 2] << std::endl;
			std::cout << std::endl;*/
			Efd_temp[i].x = sqrt(Efd[i * 3] * Efd[i * 3] + Efd_img[i * 3] * Efd_img[i * 3])*((Efd[i*3]>0)-(Efd[i*3]<0));
			Efd_temp[i].y = sqrt(Efd[i * 3 + 1] * Efd[i * 3 + 1] + Efd_img[i * 3 + 1] * Efd_img[i * 3 + 1]) * ((Efd[i * 3 + 1]>0) - (Efd[i * 3 + 1]<0));
			Efd_temp[i].z = sqrt(Efd[i * 3 + 2] * Efd[i * 3 + 2] + Efd_img[i * 3 + 2] * Efd_img[i * 3 + 2]) * ((Efd[i * 3 + 2]>0) - (Efd[i * 3 + 2]<0));
			Bfd_temp[i].x = sqrt(Bfd[i * 3] * Bfd[i * 3] + Bfd_img[i * 3] * Bfd_img[i * 3])*((Bfd_img[i * 3]<0) - (Bfd_img[i * 3]>0));
			Bfd_temp[i].y = sqrt(Bfd[i * 3 + 1] * Bfd[i * 3 + 1] + Bfd_img[i * 3 + 1] * Bfd_img[i * 3 + 1])*((Bfd_img[i * 3 + 1]<0) - (Bfd_img[i * 3 + 1]>0));
			Bfd_temp[i].z = sqrt(Bfd[i * 3 + 2] * Bfd[i * 3 + 2] + Bfd_img[i * 3 + 2] * Bfd_img[i * 3 + 2])*((Bfd_img[i * 3 + 2]<0) - (Bfd_img[i * 3 + 2]>0));
			
		
			//std::cout << Bfd_temp[i].x << std::endl;
		}
	}
	
	//normalize the fields to the maximum efield
	double efield_max = 0;
	for (i = 0; i < nnodes; i++)
	{
		efield_max = efield_max>(Efd_temp[i].x*Efd_temp[i].x + Efd_temp[i].y*Efd_temp[i].y + Efd_temp[i].z*Efd_temp[i].z) ? efield_max : (Efd_temp[i].x*Efd_temp[i].x + Efd_temp[i].y*Efd_temp[i].y + Efd_temp[i].z*Efd_temp[i].z);
	}
	efield_max = sqrt(efield_max);

	std::cout << efield_max << std::endl;
	for (i = 0; i < nnodes; i++)
	{
		Efd_temp[i].x = Efd_temp[i].x / efield_max;
		Efd_temp[i].y = Efd_temp[i].y / efield_max;
		Efd_temp[i].z = Efd_temp[i].z / efield_max;
		Bfd_temp[i].x = Bfd_temp[i].x / efield_max;
		Bfd_temp[i].y = Bfd_temp[i].y / efield_max;
		Bfd_temp[i].z = Bfd_temp[i].y / efield_max;
	}
	///*print the normalized field*/
	/*efieldfile.open("Efield_norm.txt", std::ios::out);
	for (i = 0; i < nnodes; i++)
	{
		efieldfile << Efd_temp[i].x << "	;	" << Efd_temp[i].y << "   ;   " << Efd_temp[i].z<< std::endl;
	}
	efieldfile.close();
	bfieldfile.open("Bfield_norm.txt", std::ios::out);

	for (i = 0; i < nnodes; i++)
	{
		bfieldfile << Bfd_temp[i].x << "	;	" << Bfd_temp[i].y << "   ;   " << Bfd_temp[i].z << std::endl;
	}
	bfieldfile.close();*/

	hipMalloc(&dEfd, sizeof(double3) * nnodes);
	showcudaerror();
	hipMalloc(&dBfd, sizeof(double3) * nnodes);
	showcudaerror();

	hipMemcpy(dEfd, Efd_temp, sizeof(double3)*nnodes,hipMemcpyHostToDevice);
	showcudaerror();
	hipMemcpy(dBfd, Bfd_temp, sizeof(double3)*nnodes,hipMemcpyHostToDevice);
	showcudaerror();
	std::cout << "Field info readin complete;" << std::endl;

	initphase = (int*)malloc(sizeof(int)*N_par);
	initphase_shuffle = (int*)malloc(sizeof(int)*N_par);
	hipMalloc(&dinitphase, sizeof(int)*N_par);
	showcudaerror();

	pArgc = &argc;
	pArgv = argv;
	
#if defined(__linux__)
	setenv("DISPLAY", ":0", 0);
#endif

	printf("%s starting...\n", sSDKsample);

	if (argc > 1)
	{
		if (checkCmdLineFlag(argc, (const char **)argv, "file"))
		{
			// In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
			getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
		}
	}

	printf("\n");
	/*Now that we have node, tet and field info, we will allocate the impact related array*/
	impact = new int[N_par];
	impact_shuffle = new int[N_par];
	hipMalloc(&dimpact, N_par*sizeof(int));
	showcudaerror();
	impactenergy = new double[N_par*N_cycles * 2];
	impactenergy_shuffle = new double[N_par*N_cycles * 2];
	hipMalloc(&D_impactenergy, N_par*N_cycles * 2 * sizeof(double));
	showcudaerror();

	flag = new int[N_par];
	flag_shuffle = new int[N_par];
	hipMalloc(&dflag, N_par*sizeof(int));
	showcudaerror();
	
	Hposition = (double3*)malloc(N_par*sizeof(double3));
	Hvelocity = (double3*)malloc(N_par*sizeof(double3));
	barycentric = (double4*)malloc(sizeof(double4)*N_par);
	H_position0 = (double3*)malloc(sizeof(double3)*N_par);
	H_momentumf = (double3*)malloc(sizeof(double3)*N_par);
	Hposition_shuffle = (double3*)malloc(N_par*sizeof(double3));
	Hvelocity_shuffle = (double3*)malloc(N_par*sizeof(double3));
	barycentric_shuffle = (double4*)malloc(sizeof(double4)*N_par);
	H_position0_shuffle = (double3*)malloc(sizeof(double3)*N_par);
	H_momentumf_shuffle = (double3*)malloc(sizeof(double3)*N_par);

	hipMalloc(&dbarycentric, N_par*sizeof(double4));
	hipMalloc(&d_position0, N_par*sizeof(double3));
	hipMalloc(&d_momentumf, N_par*sizeof(double3));
	hipMalloc(&D_momentumt, N_par*sizeof(double3));
	
	hipMalloc(&D_p_Efd, N_par*sizeof(double3));
	hipMalloc(&D_p_Bfd, N_par*sizeof(double3));
	//hipMalloc(&D_p_nodes, N_par*sizeof(int4));
	
	showcudaerror();

	i = 0;
	j = 0;
	k = 0;
	runTest(argc, argv, ref_file);

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();
	printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
	exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
	frameCount++;
	fpsCount++;

	if (fpsCount == fpsLimit)
	{
		avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		fpsCount = 0;
		fpsLimit = (int)MAX(avgFPS, 1.f);

		sdkResetTimer(&timer);
	}

	char fps[256];
	sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 60Hz); Master Clock Time: %3.10f (unit: # of Cycles);N_par:%d", avgFPS, g_fAnim,N_par);
	glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutInitWindowPosition(0, 0);
	glutCreateWindow("Cuda GL Interop (VBO)");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMotionFunc(motion);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	// initialize necessary OpenGL extensions
	glewInit();

	if (!glewIsSupported("GL_VERSION_2_0 "))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.01, 100.0);

	SDK_CHECK_ERROR_GL();

	return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
	// Create the CUTIL timer
	sdkCreateTimer(&timer);

	// command line mode only
	if (ref_file != NULL)
	{
		// This will pick the best possible CUDA capable device
		int devID = findCudaDevice(argc, (const char **)argv);
		// create VBO
		checkCudaErrors(hipMalloc((void **)&d_vbo_buffer, N_par * 3 * sizeof(double)));
		checkCudaErrors(hipMalloc((void **)&d_vbov_buffer, N_par * 3 * sizeof(double)));
		// run the cuda part
		runAutoTest(devID, argv, ref_file);
		// check result of Cuda step
		checkResultCuda(argc, argv, vbo);
		hipFree(d_vbo_buffer);
		hipFree(d_vbov_buffer);
		d_vbo_buffer = NULL;
		d_vbov_buffer = NULL;
	}
	else
	{
		// First initialize OpenGL context, so we can properly set the GL for CUDA.
		// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
		if (false == initGL(&argc, argv))
		{
			return false;
		}
		// use command-line specified CUDA device, otherwise use device with highest Gflops/s
		if (checkCmdLineFlag(argc, (const char **)argv, "device"))
		{
			if (gpuGLDeviceInit(argc, (const char **)argv) == -1)
			{
				return false;
			}
		}
		else
		{
			cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
		}
		// register callbacks
		glutDisplayFunc(display);
		glutKeyboardFunc(keyboard);
		glutMouseFunc(mouse);
		glutMotionFunc(motion);
		// create VBO
		createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsNone);
		createVBO(&vbov, &cuda_vbov_resource, cudaGraphicsMapFlagsNone);
		// run the cuda part
		runCuda(&cuda_vbo_resource, &cuda_vbov_resource, 0);
		std::cout << "Initalize particles finish" << std::endl;
/*
		runCuda(&cuda_vbo_resource, &cuda_vbov_resource, 1);
		hipDeviceReset();
		exit(0);
		*/

		glutMainLoop();
	}
	return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource, struct hipGraphicsResource **vbov_resource, int func)
{
	// map OpenGL buffer object for writing from CUDA
	double3 *pptr;
	double3 *vptr;
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
	size_t num_bytes;

	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&pptr, &num_bytes,*vbo_resource));
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
	checkCudaErrors(hipGraphicsMapResources(1, vbov_resource, 0));
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
	size_t num_bytesv;
	
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&vptr, &num_bytes,*vbov_resource));
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
	

	launch_kernel(pptr, vptr, g_fAnim, func);

	//hipDeviceSynchronize();

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, vbov_resource, 0));
}


void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
	printf("sdkDumpBin: <%s>\n", filename);
	FILE *fp;
	FOPEN(fp, filename, "wb");
	fwrite(data, bytes, 1, fp);
	fflush(fp);
	fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runAutoTest(int devID, char **argv, char *ref_file)
{
	char *reference_file = NULL;
	void *imageData = malloc(N_par*sizeof(double3));

	// execute the kernel
	launch_kernel((double3 *)d_vbo_buffer, (double3 *)d_vbov_buffer, g_fAnim, 0);

	//hipDeviceSynchronize();
	getLastCudaError("launch_kernel failed");

	checkCudaErrors(hipMemcpy(imageData, d_vbo_buffer, N_par*sizeof(double3), hipMemcpyDeviceToHost));

	//sdkDumpBin2(imageData, mesh_width*mesh_height*sizeof(float), "simpleGL.bin");
	reference_file = sdkFindFilePath(ref_file, argv[0]);

	if (reference_file &&
		!sdkCompareBin2BinFloat("simpleGL.bin", reference_file,
		N_par*sizeof(double3),
		MAX_EPSILON_ERROR, THRESHOLD, pArgv[0]))
	{
		g_TotalErrors++;
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
	unsigned int vbo_res_flags)
{
	assert(vbo);

	// create buffer object
	glGenBuffers(1, vbo);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);

	// initialize buffer object
	unsigned int size = N_par * 3 * sizeof(double);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

	//glBindBuffer(GL_ARRAY_BUFFER, 0);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

	// create buffer object
	glGenBuffers(1, &nvbo);
	glBindBuffer(GL_ARRAY_BUFFER, nvbo);
	// initialize buffer object
	glBufferData(GL_ARRAY_BUFFER, sizeof(double)*nnodes * 3, nodesdisp, GL_DYNAMIC_DRAW);

	// create buffer object
	glGenBuffers(1, &vboindex);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboindex);
	// initialize buffer object
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(int) * ntriangles * 3, indexes, GL_DYNAMIC_DRAW);

	// create buffer object
	glGenBuffers(1, &vboindexedg);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboindexedg);
	// initialize buffer object
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(int) * ntriangles * 6, indexesedge, GL_DYNAMIC_DRAW);

	SDK_CHECK_ERROR_GL();
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

	//// unregister this buffer object with CUDA
	//hipGraphicsUnregisterResource(vbo_res);

	//glBindBuffer(1, *vbo);
	//glDeleteBuffers(1, vbo);
	//glBindBuffer(1, nvbo);
	//glDeleteBuffers(1, &nvbo);
	//*vbo = 0;
	//nvbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
	sdkStartTimer(&timer);

	// run CUDA kernel to generate vertex positions
	runCuda(&cuda_vbo_resource, &cuda_vbov_resource, 1);
	//hipDeviceSynchronize();
	//hipStreamSynchronize(0);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(translate_x, translate_y, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);

	// render from the vbo
	glPointSize(4.0);

	glEnableClientState(GL_VERTEX_ARRAY);

	glColor3f(0.0, 0.0, 1.0);

	// create buffer object

	glBindBuffer(GL_ARRAY_BUFFER, nvbo);
	glVertexPointer(3, GL_DOUBLE, 0, 0);

	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboindex);
	//draw the surfaces
	glDrawElements(GL_TRIANGLES, ntriangles * 3, GL_UNSIGNED_INT, 0);
	////draw the vertexes
	//glPointSize(5.0);
	//glColor3f(0.0, 1.0, 0.0);
	//glDrawArrays(GL_POINTS, 0, nnodes);

	//draw the edges
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboindexedg);
	glColor3f(1.0, 0.0, 1.0);
	glDrawElements(GL_LINES, ntriangles * 6, GL_UNSIGNED_INT, 0);


	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(3, GL_DOUBLE, 0, 0);
	glColor3f(1.0, 0.0, 0.0);
	glDrawArrays(GL_POINTS, 0, N_par);
	glDisableClientState(GL_VERTEX_ARRAY);

	glutSwapBuffers();

	g_fAnim += dt;

	sdkStopTimer(&timer);
	computeFPS();
	
	if (g_fAnim >= N_cycles)
	{
		int i, j;
		
		g_fAnim = 0;
		
		if (fdnorm > fdnorm_max)
		{
			exit(3);
		}
		hipMemcpy(flag, dflag, sizeof(int)*N_par, hipMemcpyDeviceToHost);
		hipMemcpy(impactenergy, D_impactenergy, sizeof(double)*N_par*N_cycles*2, hipMemcpyDeviceToHost);
		hipMemcpy(impact, dimpact, sizeof(int)*N_par, hipMemcpyDeviceToHost);
		hipMemcpy(Hposition, d_vbo_buffer, sizeof(double3)*N_par, hipMemcpyDeviceToHost);
		std::ofstream flagfile("c:/resultsfortracking/scan/"+ std::to_string((int)fdnorm) + "flag" + ".bin", std::ios::binary);
		std::ofstream energyfile("c:/resultsfortracking/scan/" +  std::to_string((int)fdnorm) + "momentum" + ".bin", std::ios::binary);
		std::ofstream impactfile("c:/resultsfortracking/scan/" +  std::to_string((int)fdnorm) + "impact" + ".bin", std::ios::binary);
		std::ofstream positionfile("c:/resultsfortracking/scan/" + std::to_string((int)fdnorm) + "lastposition" + ".bin", std::ios::binary);
		/*flagfile.write((const char*)flag, N_par*sizeof(int));
		impactfile.write((const char*)impact, N_par*sizeof(int));
		energyfile.write((const char*)impactenergy, N_par*sizeof(double)*N_cycles*2);*/
		for (i = 0; i < N_par; i++)
		{
			positionfile << Hposition[i].x*norm << "  " << Hposition[i].y*norm << "  " << Hposition[i].z*norm << std::endl;

			flagfile << "ID="<<i<<": "<<flag[i] << std::endl;
			impactfile << "ID=" << i << ": " << impact[i] << std::endl;	
			energyfile << "ID = " << i << ":" << std::endl;
			for (j = 0; j < N_cycles * 2; j++)
			{
				energyfile << impactenergy[i*N_cycles * 2 + j] << std::endl;
			}
		}
		flagfile.close();
		impactfile.close();
		energyfile.close();
		fdnorm += fdnorm_step;// *(initphase / 100);
		activetet(nodes, xrange, yrange, zrange, norm, tetall, meshindextet_temp, &N_par, nall, tetsize);
		runCuda(&cuda_vbo_resource, &cuda_vbov_resource, 0);//reset the particles location, momentums, flags and impactmomentums.
	}
}

void timerEvent(int value)
{
	glutPostRedisplay();
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
}

void cleanup()
{
	sdkDeleteTimer(&timer);

	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}
	if (nvbo)
	{
		deleteVBO(&nvbo, cuda_vbo_resource);
	}
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27) :
		exit(EXIT_SUCCESS);
		break;
	case('r') :						//R - reset view
		/*rotate_x = 0;
		rotate_y = 0;
		translate_z = -3.0;*/
		runCuda(&cuda_vbo_resource, &cuda_vbov_resource, 0);
		break;
	case('a') :						//a - increase field level
		fdnorm += 1.0e4;
		
		printf("Max Efield = %2.3e V/m \n", fdnorm);
		break;
	case('s') :						//s - decrease field level
		fdnorm -= 1.0e4;
		
		printf("Max Efield = %2.3e V/m \n", fdnorm);
		break;
	case('d') :						//d - shut down field level
		fdnorm = 0.0;
		
		printf("Max Efield = %2.3e V/m \n", fdnorm);
		break;
	case('q') :						//q - decrease dt
		dt += 1.0e-3;
		printf("dt = %2.3f * T\n", dt);
		break;
	case('w') :						//w - decrease dt
		dt -= 1.0e-3;
		printf("dt = %2.3f * T\n", dt);
		break;
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		mouse_buttons |= 1 << button;
	}
	else if (state == GLUT_UP)
	{
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
}

void motion(int x, int y)
{
	float dx, dy;
	dx = (float)(x - mouse_old_x);
	dy = (float)(y - mouse_old_y);

	if (mouse_buttons & 1)
	{
		rotate_x += dy * 0.1f;
		rotate_y += dx * 0.1f;
	}
	else if (mouse_buttons & 4)
	{
		translate_z += dy * 0.001f;
	}
	else if (mouse_buttons & 2)
	{
		translate_x += dx*0.0005f;
		translate_y -= dy*0.0005f;
	}
	mouse_old_x = x;
	mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo)
{
	if (!d_vbo_buffer)
	{
		hipGraphicsUnregisterResource(cuda_vbo_resource);

		// map buffer object
		glBindBuffer(GL_ARRAY_BUFFER_ARB, vbo);
		float *data = (float *)glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

		// check result
		if (checkCmdLineFlag(argc, (const char **)argv, "regression"))
		{
			// write file for regression test
			/*sdkWriteFile<float>("./data/regression.dat",
				data, mesh_width * mesh_height * 3, 0.0, false);*/
		}

		// unmap GL buffer object
		if (!glUnmapBuffer(GL_ARRAY_BUFFER))
		{
			fprintf(stderr, "Unmap buffer failed.\n");
			fflush(stderr);
		}

		checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo,
			cudaGraphicsMapFlagsWriteDiscard));

		SDK_CHECK_ERROR_GL();
	}
}
